
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 512

__device__ unsigned int warpDistribution[33] = {0};

__device__ void countWarpDistribution(){

      unsigned int mask = __popc(__activemask());
      if(threadIdx.x % 32 == 0){
        atomicAdd(&warpDistribution[mask],1);
      }
}

__device__ void printWarpDistribution(){
    printf("\n Warp Distribution: \n");
    for(int i = 0; i < 33; i++){
        printf("W%d: %u, ",i,warpDistribution[i]);
    }
    printf("\n\n");
}

__global__ void naiveReduction(float *out, float *in, unsigned size)
{
    
    __shared__ float partialSum[2*BLOCK_SIZE];
    unsigned int t = threadIdx.x;
    
    unsigned int start= 2*blockIdx.x*blockDim.x;
    if((start + t)>size){
       partialSum[t] =  0;
    }
    else{
    partialSum[t] =in[start + t];
    }
    if((start + t + blockDim.x)>size){
      partialSum[t+blockDim.x] = 0;
    }
    else{
    partialSum[t+blockDim.x] =in[start + t + blockDim.x];
    }
    for(unsigned int stride = 1; stride<= blockDim.x;stride*=2){
      __syncthreads();
      if(t % stride == 0){
        partialSum[2*t] += partialSum[2*t+stride];
      }

    }
    
      out[t*blockDim.x+blockIdx.x] =partialSum[2*t];
 
}


__global__ void optimizedReduction(float *out, float *in, unsigned size){

   __shared__ float partialSum[2*BLOCK_SIZE];
    unsigned int t = threadIdx.x;
    
    unsigned int start= 2*blockIdx.x*blockDim.x;
    if((start + t)>size){
       partialSum[t] =  0;
    }
    else{
    partialSum[t] =in[start + t];
    }
    if((start + t + blockDim.x)>size){
      partialSum[t+blockDim.x] = 0;
    }
    else{
    partialSum[t+blockDim.x] =in[start + t + blockDim.x];
    }
    for(unsigned int stride = blockDim.x; stride>0 ;stride/=2){
      __syncthreads();
      if(t < stride ){
        partialSum[t] += partialSum[t+stride];
         
      }

    }
    
      out[t*blockDim.x+blockIdx.x] =partialSum[t];


  
}